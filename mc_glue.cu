#include "mc_glue.h"
#include "cuda_func.h"

void mc_glue::cuda_init()
{


    hipFree(d_a2fVertexOffset_);
    CHECK(hipMalloc((float**)&d_a2fVertexOffset_,sizeof(float)*8*3));
    hipMemcpy(d_a2fVertexOffset_,a2fVertexOffset,sizeof(float)*8*3,hipMemcpyHostToDevice);
    
    hipFree(d_a2iEdgeConnection_);
    CHECK(hipMalloc((int**)&d_a2iEdgeConnection_,sizeof(int)*12*2));
    hipMemcpy(d_a2iEdgeConnection_,a2iEdgeConnection,sizeof(int)*12*2,hipMemcpyHostToDevice);
    
    hipFree(d_a2fEdgeDirection_);
    CHECK(hipMalloc((float**)&d_a2fEdgeDirection_,sizeof(float)*12*3));
    hipMemcpy(d_a2fEdgeDirection_,a2fEdgeDirection,sizeof(float)*12*3,hipMemcpyHostToDevice);
    
    hipFree(d_aiCubeEdgeFlags_);
    CHECK(hipMalloc((int**)&d_aiCubeEdgeFlags_,sizeof(int)*256));
    hipMemcpy(d_aiCubeEdgeFlags_,aiCubeEdgeFlags,sizeof(int)*256,hipMemcpyHostToDevice);
    
    hipFree(d_a2iTriangleConnectionTable_);
    CHECK(hipMalloc((int**)&d_a2iTriangleConnectionTable_,sizeof(int)*256*16));
    hipMemcpy(d_a2iTriangleConnectionTable_,a2iTriangleConnectionTable,sizeof(int)*256*16,hipMemcpyHostToDevice);
    
    hipFree(d_number_record_);
    CHECK(hipMalloc((int**)&d_number_record_,sizeof(int)*2));
    hipMemcpy(d_number_record_,number_record_,sizeof(int)*2,hipMemcpyHostToDevice);
    
}


void mc_glue::cuda_get_sdf_values()
{
    hipFree(d_sdf_);
    CHECK(hipMalloc((float**)&d_sdf_,sizeof(float)*NX*NY*NZ));
    hipMemcpy(d_sdf_,sdf_readin_,sizeof(float)*NX*NY*NZ,hipMemcpyHostToDevice);
    
    hipFree(d_edge_point_state_);
    CHECK(hipMalloc((int**)&d_edge_point_state_,sizeof(int)*NX*NY*NZ*3));
    hipMemcpy(d_edge_point_state_,edge_point_state,sizeof(int)*NX*NY*NZ*3,hipMemcpyHostToDevice);
    
    
    hipFree(d_points_coor_);
    CHECK(hipMalloc((float**)&d_points_coor_,sizeof(float)*3*(int)(NX*NY*NZ*12*0.05)));
    
    hipFree(d_faces_index_);
    CHECK(hipMalloc((int**)&d_faces_index_,sizeof(int)*3*(int)(NX*NY*NZ*5*0.05)));
    
    hipFree(d_faces_ijkd_);
    CHECK(hipMalloc((int**)&d_faces_ijkd_,sizeof(int)*3*4*(int)(NX*NY*NZ*5*0.05)));
}


void mc_glue::mc_get_mesh_on_gpu()
{
    int n_thread=NX*NY*NZ;
    d_mc_get_mesh_on_gpu<<<GET_BLOCKS(n_thread),CUDA_NUM_THREADS>>>(n_thread,d_sdf_,d_edge_point_state_,NX,NY,NZ, fTargetValue, d_number_record_,d_a2fVertexOffset_,d_a2iEdgeConnection_,d_a2fEdgeDirection_,d_aiCubeEdgeFlags_,d_a2iTriangleConnectionTable_,d_points_coor_,d_faces_ijkd_,d_faces_index_);
}


void mc_glue::convert_ijkd_to_pindex_on_gpu()
{
    int n_thread=number_record_[1];
    d_conver_ijkd_to_pindex<<<GET_BLOCKS(n_thread),CUDA_NUM_THREADS>>>(n_thread,NX,NY,NZ, d_edge_point_state_, d_faces_ijkd_, d_faces_index_, d_number_record_);
}

